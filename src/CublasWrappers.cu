// SPDX-FileCopyrightText: 2023 CERN
// SPDX-License-Identifier: Apache-2.0

#include <cub/device/device_merge_sort.cuh>
#include <AdePT/core/CublasWrappers.cuh>
#include <AdePT/core/ScoringCommons.hh>
#include <stdio.h>

// template <>
// hipError_t hipcub::DeviceMergeSort::SortKeys(
//     void*,
//     std::size_t&,
//     GPUHit*,
//     unsigned int,
//     AsyncAdePT::CompareGPUHits,
//     hipStream_t);

namespace cublas_wrappers {
template <typename KeyIteratorT, typename OffsetT, typename CompareOpT>
hipError_t CublasSortKeys(void *d_temp_storage, std::size_t &temp_storage_bytes, KeyIteratorT d_keys,
                           OffsetT num_items, CompareOpT compare_op, hipStream_t stream)
{
  printf("CublasSortKeys Called\n");
  return hipError_t{};
}

// Explicit instantiations of these templates
template hipError_t CublasSortKeys<GPUHit*, unsigned int, CompareGPUHits>
                                        (void*, 
                                        unsigned long&, 
                                        GPUHit*, 
                                        unsigned int, 
                                        CompareGPUHits, 
                                        ihipStream_t*);

} // namespace cublas_wrappers
