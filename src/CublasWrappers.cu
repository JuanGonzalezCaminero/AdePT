// SPDX-FileCopyrightText: 2023 CERN
// SPDX-License-Identifier: Apache-2.0

#include <cub/device/device_merge_sort.cuh>
#include <AdePT/core/CublasWrappers.cuh>
#include <AdePT/core/ScoringCommons.hh>
#include <stdio.h>

// template <>
// hipError_t hipcub::DeviceMergeSort::SortKeys(
//     void*,
//     std::size_t&,
//     GPUHit*,
//     unsigned int,
//     AsyncAdePT::CompareGPUHits,
//     hipStream_t);

namespace cublas_wrappers {
template <typename KeyIteratorT, typename OffsetT, typename CompareOpT>
hipError_t CublasSortKeys(void *d_temp_storage, std::size_t &temp_storage_bytes, KeyIteratorT d_keys,
                           OffsetT num_items, CompareOpT compare_op, hipStream_t stream)
{
    return hipcub::DeviceMergeSort::SortKeys(d_temp_storage, 
                                        temp_storage_bytes, 
                                        d_keys, 
                                        num_items,
                                        compare_op,
                                        stream);
//   printf("CublasSortKeys Called\n");
//   return hipError_t{};
}

// Explicit instantiations of these templates

// From HitScoring::HitScoring
template hipError_t CublasSortKeys<GPUHit*, unsigned int, CompareGPUHits>
                                        (void*, 
                                        unsigned long&, 
                                        GPUHit*, 
                                        unsigned int,
                                        CompareGPUHits, 
                                        ihipStream_t*);

} // namespace cublas_wrappers
