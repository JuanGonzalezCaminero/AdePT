// SPDX-FileCopyrightText: 2023 CERN
// SPDX-License-Identifier: Apache-2.0

#include <cub/device/device_merge_sort.cuh>
#include <AdePT/core/CublasWrappers.cuh>
#include <AdePT/core/ScoringCommons.hh>
#include <stdio.h>

// template <>
// hipError_t hipcub::DeviceMergeSort::SortKeys(
//     void*,
//     std::size_t&,
//     GPUHit*,
//     unsigned int,
//     AsyncAdePT::CompareGPUHits,
//     hipStream_t);

namespace cublas_wrappers {
template <typename KeyIteratorT, typename OffsetT, typename CompareOpT>
hipError_t CublasSortKeys(void *d_temp_storage, std::size_t &temp_storage_bytes, KeyIteratorT d_keys,
                           OffsetT num_items, CompareOpT compare_op, hipStream_t stream)
{
    printf("Cublas sort called\n");
    return hipcub::DeviceMergeSort::SortKeys(d_temp_storage, 
                                          temp_storage_bytes, 
                                          d_keys, 
                                          num_items,
                                          compare_op,
                                          stream);
}

// Explicit instantiations of these templates

// From HitScoring::HitScoring
template hipError_t CublasSortKeys<GPUHit*, unsigned int, CompareGPUHits>
                                        (void*, 
                                        std::size_t &, 
                                        GPUHit*, 
                                        unsigned int,
                                        CompareGPUHits, 
                                        hipStream_t);

} // namespace cublas_wrappers
